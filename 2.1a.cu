#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be foundnvcc 
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o cudapass90 cudapass90.cu
    ./cudapass90 
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char plain_password1[] = "BI1456";
	char plain_password2[] = "KE2378";
	char plain_password3[] = "SH3712";
	char plain_password4[] = "RA9222";


	char *aa = attempt;
	char *bb = attempt;
	char *cc = attempt;
	char *dd = attempt;
	char *aa1 = plain_password1;
	char *bb2 = plain_password2;
	char *cc3 = plain_password3;
	char *dd4 = plain_password4;

	while(*aa == *aa1) { 
		if(*aa == '\0') 
		{
			printf("Password: %s\n",plain_password1);
			break;
		}

		aa++;
		aa1++;
	}
	
	while(*bb == *bb2) { 
		if(*bb == '\0') 
		{
			printf("Password: %s\n",plain_password2);
			break;
		}

		bb++;
		bb2++;
	}

	while(*cc == *cc3) { 
		if(*cc == '\0') 
		{
			printf("Password: %s\n",plain_password3);
			break;
		}

		cc++;
		cc3++;
	}

	while(*dd == *dd4) { 
		if(*dd == '\0') 
		{
			printf("Password: %s\n",plain_password4);
			return 1;
		}

		dd++;
		dd4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

// Calculate the difference between two times. Returns zero on
// success and the time difference through an argument. It will 
// be unsuccessful if the start time is after the end time.

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int d_sec =  finish->tv_sec - start->tv_sec; 
  long long int d_nsec =  finish->tv_nsec - start->tv_nsec; 

  if(d_nsec < 0 ) {
    d_sec--;
    d_nsec += 1000000000; 
  } 
  *difference = d_sec * 1000000000 + d_nsec;
  return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


